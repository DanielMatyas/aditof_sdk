#include "hip/hip_runtime.h"
/*
 * BSD 3-Clause License
 *
 * Copyright (c) 2019, Analog Devices, Inc.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 *    list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 *    contributors may be used to endorse or promote products derived from
 *    this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "../include/cuda_utils.h"
#include <assert.h>
#include <iostream>
#include <stdio.h>

#include "data.h"
#include "npy.hpp"

#define INPUT_WIDTH 40
#define INPUT_HEIGHT 30

#define FRAME_WIDTH 640
#define FRAME_HEIGHT 480

#define MAX_FRAME_VALUE 800
#define SUBFRAME_NUMBER 1245

namespace aditof {};

// -------------------      CUDA        -----------------------------------------------

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA_CXX: DA Runtime Error: %s\n",
                hipGetErrorString(result));
        assert(result == hipSuccess);
    }
#endif
    return result;
}

__global__ void buildDistortionCorrectionCacheCuda(double *m_distortion_cache_d,
                                                   double *m_parameters_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    int i = threadPosition % (int)m_parameters_d[0];
    int j = threadPosition / (int)m_parameters_d[0];

    if (i >= 0 && i < m_parameters_d[0]) {
        if (j >= 0 && j < m_parameters_d[1]) {
            double x = (i - m_parameters_d[4]) / m_parameters_d[2];
            double y = (j - m_parameters_d[5]) / m_parameters_d[3];

            double r2 = x * x + y * y;
            double k_calc = double(1 + m_parameters_d[6] * r2 +
                                   m_parameters_d[7] * r2 * r2 +
                                   m_parameters_d[8] * r2 * r2 * r2);
            m_distortion_cache_d[j * (int)m_parameters_d[0] + i] = k_calc;
        }
    }
}

__global__ void
applyDistortionCorrectionCacheCuda(uint16_t *m_frame_d, uint16_t *tmp_frame,
                                   double *m_parameters_d,
                                   double *m_distortion_cache_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    int i = threadPosition % (int)m_parameters_d[0];
    int j = threadPosition / (int)m_parameters_d[0];

    if (i >= 0 && i < m_parameters_d[0]) {
        if (j >= 0 && j < m_parameters_d[1]) {

            double x = (double(i) - m_parameters_d[4]) / m_parameters_d[2];
            double y = (double(j) - m_parameters_d[5]) / m_parameters_d[3];

            //apply correction
            double x_dist_adim =
                x * m_distortion_cache_d[j * (int)m_parameters_d[0] + i];
            double y_dist_adim =
                y * m_distortion_cache_d[j * (int)m_parameters_d[0] + i];

            //back to original space
            int x_dist =
                (int)(x_dist_adim * m_parameters_d[2] + m_parameters_d[4]);
            int y_dist =
                (int)(y_dist_adim * m_parameters_d[3] + m_parameters_d[5]);

            if (x_dist >= 0 && x_dist < (int)m_parameters_d[0] && y_dist >= 0 &&
                y_dist < (int)m_parameters_d[1]) {
                m_frame_d[j * (int)m_parameters_d[0] + i] =
                    tmp_frame[y_dist * (int)m_parameters_d[0] + x_dist];
            } else {
                m_frame_d[j * (int)m_parameters_d[0] + i] =
                    tmp_frame[j * (int)m_parameters_d[0] + i];
            }
        }
    }
}

__global__ void buildGeometryCorrectionCacheCuda(double *m_geometry_cache_d,
                                                 double *m_parameters_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    int i = threadPosition / (int)m_parameters_d[0];
    int j = threadPosition % (int)m_parameters_d[0];

    if (i >= 0 && i < m_parameters_d[1]) {
        if (j >= 0 && j < m_parameters_d[0]) {
            double tanXAngle = (m_parameters_d[9] - j) / m_parameters_d[2];
            double tanYAngle = (m_parameters_d[10] - i) / m_parameters_d[3];

            m_geometry_cache_d[i * (int)m_parameters_d[0] + j] =
                1.0 / sqrt(1 + tanXAngle * tanXAngle + tanYAngle * tanYAngle);
        }
    }
}

__global__ void applyGeometryCorrectionCacheCuda(uint16_t *m_frame_d,
                                                 double *m_parameters_d,
                                                 double *m_geometry_cache_d

) {
    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    if (threadPosition >= 0 &&
        threadPosition < m_parameters_d[0] * m_parameters_d[1]) {
        if (m_frame_d[threadPosition] > m_parameters_d[14])
            m_frame_d[threadPosition] = m_parameters_d[14];
        else
            m_frame_d[threadPosition] =
                m_frame_d[threadPosition] * m_geometry_cache_d[threadPosition];
    }
}

__global__ void buildDepthCorrectionCacheCuda(uint16_t *m_depth_cache_d,
                                              double *m_parameters_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;
    if (threadPosition < m_parameters_d[13]) {
        int16_t currentValue = static_cast<int16_t>(
            static_cast<float>(threadPosition) * m_parameters_d[11] +
            m_parameters_d[12]);
        m_depth_cache_d[threadPosition] = currentValue <= m_parameters_d[14]
                                              ? currentValue
                                              : m_parameters_d[14];
    }
}

__global__ void applyDepthCorrectionCacheCuda(uint16_t *m_frame_d,
                                              double *m_parameters_d,
                                              uint16_t *m_depth_cache_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    if (threadPosition >= 0 &&
        threadPosition < m_parameters_d[0] * m_parameters_d[1]) {
        *(m_frame_d + threadPosition) =
            *(m_depth_cache_d + *(m_frame_d + threadPosition));
    }
}

//--------------------------    CLASS   -------------------------------------------------------------

void cudaOnTarget::buildDistortionCorrectionCache() {

    std::cout << "CUDA_CXX: Building Distortion correction\n";

    checkCuda(hipMalloc((void **)&m_distortion_cache_d,
                         sizeof(double) * m_parameters[0] * m_parameters[1]));

    buildDistortionCorrectionCacheCuda<<<m_parameters[0] * m_parameters[1] /
                                             THREAD_PER_BLOCK,
                                         THREAD_PER_BLOCK>>>(
        m_distortion_cache_d, m_parameters_d);

    // m_distortion_cache =
    //     (double *)malloc(sizeof(double) * m_parameters[0] * m_parameters[1]);
    // checkCuda(hipMemcpy(m_distortion_cache, m_distortion_cache_d,
    //            sizeof(double) * m_parameters[0] * m_parameters[1],
    //            hipMemcpyDeviceToHost));

    // std::cout << "GPU distortion: \n";
    // for (int i = 0; i < 10; i++) {
    //     std::cout << m_distortion_cache[i] << ", ";
    // }
    // std::cout << "\n\n\n";
}

void cudaOnTarget::buildGeometryCorrectionCache() {

    std::cout << "CUDA_CXX: Building Geometry correction\n";

    checkCuda(hipMalloc((void **)&m_geometry_cache_d,
                         sizeof(double) * m_parameters[0] * m_parameters[1]));

    //Check if more blocks nedded than resulted from division
    int nrOfBlocks =
        ((m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK) *
             THREAD_PER_BLOCK <
         m_parameters[0] * m_parameters[1])
            ? m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK + 1
            : m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK;
    buildGeometryCorrectionCacheCuda<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
        m_geometry_cache_d, m_parameters_d);

    // m_geometry_cache =
    //     (double *)malloc(sizeof(double) * m_parameters[0] * m_parameters[1]);
    // checkCuda(hipMemcpy(m_geometry_cache, m_geometry_cache_d,
    //            sizeof(double) * m_parameters[0] * m_parameters[1],
    //            hipMemcpyDeviceToHost));

    // std::cout << "GPU geometry: \n";
    // for (int i = 0; i < 10; i++) {
    //     std::cout << m_geometry_cache[i] << ", ";
    // }
    // std::cout << "\n\n\n";
}

void cudaOnTarget::buildDepthCorrectionCache() {

    std::cout << "CUDA_CXX: Building Depth correction\n";

    checkCuda(hipMalloc((void **)&m_depth_cache_d,
                         sizeof(uint16_t) * m_parameters[13]));

    //Check if more blocks nedded than resulted from division
    int nrOfBlocks = ((m_parameters[13] / THREAD_PER_BLOCK) * THREAD_PER_BLOCK <
                      m_parameters[13])
                         ? m_parameters[13] / THREAD_PER_BLOCK + 1
                         : m_parameters[13] / THREAD_PER_BLOCK;
    buildDepthCorrectionCacheCuda<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
        m_depth_cache_d, m_parameters_d);

    // m_depth_cache = (uint16_t *)malloc(sizeof(uint16_t) * m_parameters[13]);
    // checkCuda(hipMemcpy(m_depth_cache, m_depth_cache_d,
    //    sizeof(uint16_t) * m_parameters[13], hipMemcpyDeviceToHost));

    // std::cout << "GPU depth: \n";
    // for (int i = 0; i < 10; i++) {
    //     std::cout << m_depth_cache[i] << ", ";
    // }
    // std::cout << "\n\n\n";
}

void cudaOnTarget::applyDistortionCorrection() {

    //create temporary frame buffer
    uint16_t *tmp_frame;
    checkCuda(hipMalloc((void **)&tmp_frame,
                         sizeof(uint16_t) * m_parameters[0] * m_parameters[1]));
    checkCuda(hipMemcpy(tmp_frame, m_frame_d,
                         sizeof(uint16_t) * m_parameters[0] * m_parameters[1],
                         hipMemcpyDeviceToDevice));

    //Check if more blocks nedded than resulted from division
    int nrOfBlocks =
        ((m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK) *
             THREAD_PER_BLOCK <
         m_parameters[0] * m_parameters[1])
            ? m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK + 1
            : m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK;
    applyDistortionCorrectionCacheCuda<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
        m_frame_d, tmp_frame, m_parameters_d, m_distortion_cache_d);
    checkCuda(hipFree(tmp_frame));
}
void cudaOnTarget::applyDepthCorrection() {

    //Check if more blocks nedded than resulted from division
    int nrOfBlocks =
        ((m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK) *
             THREAD_PER_BLOCK <
         m_parameters[0] * m_parameters[1])
            ? m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK + 1
            : m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK;
    applyDepthCorrectionCacheCuda<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
        m_frame_d, m_parameters_d, m_depth_cache_d);
}

void cudaOnTarget::applyGeometryCorrection() {
    //Check if more blocks nedded than resulted from division
    int nrOfBlocks =
        ((m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK) *
             THREAD_PER_BLOCK <
         m_parameters[0] * m_parameters[1])
            ? m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK + 1
            : m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK;
    applyGeometryCorrectionCacheCuda<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
        m_frame_d, m_parameters_d, m_geometry_cache_d);
}

void cudaOnTarget::cpyFrameToGPU(uint16_t *frame) {
    checkCuda(hipMemcpy(m_frame_d, frame,
                         sizeof(uint16_t) * m_parameters[0] * m_parameters[1],
                         hipMemcpyHostToDevice));
    memcpy(m_frame, frame, FRAME_WIDTH * FRAME_HEIGHT * sizeof(uint16_t));
}
void cudaOnTarget::cpyFrameFromGPU(uint16_t *frame) {
    checkCuda(hipMemcpy(frame, m_frame_d,
                         sizeof(uint16_t) * m_parameters[0] * m_parameters[1],
                         hipMemcpyDeviceToHost));
}

void cudaOnTarget::printFrameFromGPU() {
    cpyFrameFromGPU(m_frame);
    for (int i = 0; i < 10; i++) {
        std::cout << m_frame[i] << ", ";
    }
}

void cudaOnTarget::setParameters(double width, double height, double fx,
                                 double fy, double cx, double cy, double k1,
                                 double k2, double k3, double x0, double y0,
                                 double gain, double offset,
                                 double pixelMaxValue, double range) {
    //Moving parameters on GPU memory
    double parameters[15] = {width, height, fx, fy, cx,   cy,     k1,
                             k2,    k3,     x0, y0, gain, offset, pixelMaxValue,
                             range};
    m_parameters = (double *)malloc(15 * sizeof(double));
    memcpy(m_parameters, parameters, 15 * sizeof(double));

    checkCuda(hipMalloc((void **)&m_parameters_d, sizeof(double) * 15));
    checkCuda(hipMemcpy(m_parameters_d, parameters, sizeof(double) * 15,
                         hipMemcpyHostToDevice));

    //allocating memory for frame
    checkCuda(
        hipMalloc((void **)&m_frame_d, sizeof(uint16_t) * width * height));
    m_frame = (uint16_t *)malloc(sizeof(uint16_t) * width * height);

    //load neural network model
    loadNetworkModel();
}

void cudaOnTarget::freeAll() {
    checkCuda(hipFree(m_geometry_cache_d));
    checkCuda(hipFree(m_distortion_cache_d));
    checkCuda(hipFree(m_depth_cache_d));
    checkCuda(hipFree(m_frame_d));
    checkCuda(hipFree(m_parameters_d));
    checkCuda(hipFree(m_network_d));
    checkCuda(hipFree(m_layers_d));
    checkCuda(hipFree(m_subFrameParameters_d));
    checkCuda(hipFree(m_subFrameOutputs_d));
}

std::string cudaOnTarget::getFileNameWeights(std::string fileName) {
    std::string firstPart = PATH_TO_CNN_JSON;
    std::string lastPartWeights = "_weights.txt";

    return (firstPart.append(fileName.append(lastPartWeights)));
}
std::string cudaOnTarget::getFileNameBias(std::string fileName) {
    std::string firstPart = PATH_TO_CNN_JSON;
    std::string lastPartBias = "_bias.txt";

    return (firstPart.append(fileName.append(lastPartBias)));
}

void cudaOnTarget::readInLayer(std::vector<Layer> &network,
                               std::string fileName) {
    std::string fileNameWeights = getFileNameWeights(fileName);
    std::string fileNameBias = getFileNameBias(fileName);

    std::ifstream myFileWeights; // creates stream myFile
    std::ifstream myFileBias;    // creates stream myFile

    myFileWeights.open(fileNameWeights); // opens .txt file
    myFileBias.open(fileNameBias);       // opens .txt file
    if (!myFileWeights.is_open() ||
        !myFileBias.is_open()) // check file is open, quit if not
    {
        std::cerr << "failed to open file\n";
        return;
    }

    std::vector<double> weights; // vector to store the numerical values in
    std::vector<double> bias;    // vector to store the numerical values in

    double number = 0;
    while (myFileWeights >> number) {
        weights.push_back(number);
    }
    while (myFileBias >> number) {
        bias.push_back(number);
    }

    Layer layer;
    layer.name = fileName;
    layer.weights = weights;
    layer.bias = bias;

    network.push_back(layer);

    std::cout << "########\nWeights: " << weights.size()
              << "\nBiases: " << bias.size() << std::endl;
}

void cudaOnTarget::loadNetworkModel() {

    readInLayer(Network, "layer_0");
    readInLayer(Network, "layer_1");
    readInLayer(Network, "layer_2");
    readInLayer(Network, "layer_3");

    cpyNetworkToGPU();
    loadNetworkParameters();
}

void cudaOnTarget::cpyNetworkToGPU() {
    if (Network.size() == 0) {
        std::cout << "CUDA_CXX: Please load the model first!\n";
        return;
    } else {
        int sizeNetworkTmp = 1;
        int sizeLayersTmp = 0;
        sizeNetworkTmp += Network.size();
        for (int i = 0; i < Network.size(); i++) {
            sizeNetworkTmp += Network[i].weights.size();
            sizeNetworkTmp += Network[i].bias.size();
            sizeLayersTmp += Network[i].bias.size();
        }
        //allocate memory for network
        checkCuda(
            hipMalloc((void **)&m_network_d, sizeof(double) * sizeNetworkTmp));
        checkCuda(
            hipMalloc((void **)&m_layers_d, sizeof(double) * sizeLayersTmp));

        //aproximating at the 7th decimal value !!!

        //serialize network
        int index = 0;
        double *serialNetwork;
        serialNetwork = (double *)malloc(sizeNetworkTmp * sizeof(double));
        serialNetwork[index++] = (double)Network.size();
        for (int i = 0; i < serialNetwork[0]; i++)
            serialNetwork[index++] = Network[i].bias.size();
        for (int i = 0; i < serialNetwork[0]; i++) {
            for (int j = 0; j < Network[i].weights.size(); j++)
                serialNetwork[index++] = Network[i].weights[j];
            for (int j = 0; j < Network[i].bias.size(); j++)
                serialNetwork[index++] = Network[i].bias[j];
        }

        std::cout << "\nSerialized network legth: " << index << std::endl;

        //copy data to network
        checkCuda(hipMemcpy(m_network_d, serialNetwork,
                             sizeof(double) * sizeNetworkTmp,
                             hipMemcpyHostToDevice));
        // free(serialNetwork);

        // std::cout << "Serialized Network: \n\n";
        // for (int i = 0; i < index; i++) {
        //     std::cout << serialNetwork[i] << ", ";
        // }
        // std::cout << "\n\n";
    }
}

void cudaOnTarget::loadNetworkParameters() {


    m_subFrameParameters = (int *)malloc(sizeof(int) * SUBFRAME_NUMBER * 3);
    checkCuda(hipMalloc((void **)&m_subFrameParameters_d,
                         sizeof(int) * SUBFRAME_NUMBER * 3));

    //allocating memory for output layer calculation
    m_subFrameOutputs = (double *)malloc(sizeof(double) * SUBFRAME_NUMBER);
    checkCuda(hipMalloc((void **)&m_subFrameOutputs_d,
                         sizeof(double) * SUBFRAME_NUMBER));

    //generating subframes for different resolutions
    int poz = 0;
    for (int resolution = 1; resolution <= 16; resolution *= 2) {
        int stride_x = resolution * INPUT_WIDTH / 2;
        int stride_y = resolution * INPUT_HEIGHT / 2;
        for (int x_offset = 0;
             x_offset <= (FRAME_WIDTH - resolution * INPUT_WIDTH);
             x_offset += stride_x) {
            for (int y_offset = 0;
                 y_offset <= (FRAME_HEIGHT - resolution * INPUT_HEIGHT);
                 y_offset += stride_y) {

                //copy data to GPU memory, order: resolution, x_offset, y_offset
                m_subFrameParameters[poz++] = resolution;
                m_subFrameParameters[poz++] = x_offset;
                m_subFrameParameters[poz++] = y_offset;
            }
        }
    }

    checkCuda(hipMemcpy((void **)m_subFrameParameters_d, m_subFrameParameters,
                         sizeof(int) * SUBFRAME_NUMBER * 3,
                         hipMemcpyHostToDevice));
}

__global__ void calcNetLayer(double *inputLayer, double *inputSize,
                             double *outputLayer, double *outputSize,
                             double *weights, double *bias) {

    int poz = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    if (poz >= 0 && poz < (int)outputSize[0]) {

        *(outputLayer + poz) = 0;
        for (int i = 0; i < (int)inputSize[0]; i++) {
            *(outputLayer + poz) +=
                (*(inputLayer + i) *
                 (*(weights + poz * (int)inputSize[0] + i)));
        }
        *(outputLayer + poz) += *(bias + poz);
        *(outputLayer + poz) = 1 / (1 + exp(-1 * (*(outputLayer + poz))));
    }
}

__global__ void calcFirstNetLayer(uint16_t *frame,  double *layer,
                                  double *weights, double *bias,
                                  double *nrOfNodes, int *frameParameters) {

    int poz = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    if (poz >= 0 && poz < (int)nrOfNodes[0]) {

        *(layer + poz) = 0;
        for (int i = 0; i < INPUT_WIDTH; i++) {
            for (int j = 0; j < INPUT_HEIGHT; j++) {
                *(layer + poz) +=
                    (*(weights + poz * INPUT_WIDTH * INPUT_HEIGHT +
                       j * INPUT_WIDTH + i)) *
                    (*(frame +
                       (frameParameters[2] + j * frameParameters[0]) *
                           FRAME_WIDTH +
                       (frameParameters[1] + i * frameParameters[2])));


            }
        }
        layer[poz] += bias[poz];
        layer[poz] = 1 / (1 + exp(-1 * (layer[poz])));
    }
}

void cudaOnTarget::calculateNetworkOutput() {

    for (int subFrameParameterIndex = 0;
         subFrameParameterIndex < SUBFRAME_NUMBER;
         subFrameParameterIndex += 3) {

        int layerIndex = 0;         //for m_layers_d
        int previousLayerIndex = 0; //for m_layers_d

        int nodeNumberIndex = 1;              //for m_network_d
        int weightIndex = 1 + Network.size(); //for m_network_d
        int biasIndex =
            1 + Network.size() + Network[0].weights.size(); //for m_network_d

        //calculate first layer using frame input and subFrameParameters
        for (int i = 0; i < Network.size(); i++) {
            int nrOfBlocks = ((Network[i].bias.size() / THREAD_PER_BLOCK) *
                                  THREAD_PER_BLOCK <
                              Network[i].bias.size())
                                 ? Network[i].bias.size() / THREAD_PER_BLOCK + 1
                                 : Network[i].bias.size() / THREAD_PER_BLOCK;
            // save output of last layer in separate array
            if (i == Network.size() - 1) {
                calcNetLayer<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
                    (m_layers_d + previousLayerIndex),
                    (m_network_d + nodeNumberIndex - 1),
                    (m_subFrameOutputs_d + subFrameParameterIndex / 3),
                    (m_network_d + nodeNumberIndex),
                    (m_network_d + weightIndex), (m_network_d + biasIndex));
            }
            //first layer, optimized on input
            else if (i == 0) {
                calcFirstNetLayer<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
                    m_frame_d,
                    (m_layers_d + layerIndex), (m_network_d + weightIndex),
                    (m_network_d + biasIndex), (m_network_d + nodeNumberIndex),
                    (m_subFrameParameters_d + subFrameParameterIndex));

            }
            //intermidiate layer
            else {
                calcNetLayer<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
                    (m_layers_d + previousLayerIndex),
                    (m_network_d + nodeNumberIndex - 1),
                    (m_layers_d + layerIndex), (m_network_d + nodeNumberIndex),
                    (m_network_d + weightIndex), (m_network_d + biasIndex));
            }

            weightIndex = biasIndex + Network[i].bias.size();
            if (i < (Network.size() - 1))
                biasIndex = weightIndex + Network[i + 1].weights.size();
            previousLayerIndex = layerIndex;
            layerIndex += Network[i].bias.size();
            nodeNumberIndex++;
        }
    }

    checkCuda(hipMemcpy((void **)m_subFrameOutputs, m_subFrameOutputs_d,
                         sizeof(double) * SUBFRAME_NUMBER,
                         hipMemcpyDeviceToHost));

    
}
