#include "hip/hip_runtime.h"
/*
 * BSD 3-Clause License
 *
 * Copyright (c) 2019, Analog Devices, Inc.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 *    list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 *    contributors may be used to endorse or promote products derived from
 *    this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include "../include/cuda_utils.h"
#include <iostream>
namespace aditof {};

// -------------------      CUDA        -----------------------------------------------

__global__ void buildDistortionCorrectionCacheCuda(double *m_distortion_cache_d,
                                                   double *m_parameters_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    int i = threadPosition % (int)m_parameters_d[0];
    int j = threadPosition / (int)m_parameters_d[0];

    if (i >= 0 && i < m_parameters_d[0]) {
        if (j >= 0 && j < m_parameters_d[1]) {
            double x = (i - m_parameters_d[4]) / m_parameters_d[2];
            double y = (j - m_parameters_d[5]) / m_parameters_d[3];

            double r2 = x * x + y * y;
            double k_calc = double(1 + m_parameters_d[6] * r2 +
                                   m_parameters_d[7] * r2 * r2 +
                                   m_parameters_d[8] * r2 * r2 * r2);
            m_distortion_cache_d[j * (int)m_parameters_d[0] + i] = k_calc;
        }
    }
}

__global__ void applyDistortionCorrectionCacheCuda(double *m_distortion_cache_d,
                                                   double *m_parameters_d,
                                                   uint16_t *frame) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    int i = threadPosition % (int)m_parameters_d[0];
    int j = threadPosition / (int)m_parameters_d[0];

    if (i >= 0 && i < m_parameters_d[0]) {
        if (j >= 0 && j < m_parameters_d[1]) {
            // TO BE IMPLEMENTED
        }
    }
}

__global__ void buildGeometryCorrectionCacheCuda(double *m_geometry_cache_d,
                                                 double *m_parameters_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    int i = threadPosition / (int)m_parameters_d[0];
    int j = threadPosition % (int)m_parameters_d[0];

    if (i >= 0 && i < m_parameters_d[1]) {
        if (j >= 0 && j < m_parameters_d[0]) {
            double tanXAngle = (m_parameters_d[9] - j) / m_parameters_d[2];
            double tanYAngle = (m_parameters_d[10] - i) / m_parameters_d[3];

            m_geometry_cache_d[i * (int)m_parameters_d[0] + j] =
                1.0 / sqrt(1 + tanXAngle * tanXAngle + tanYAngle * tanYAngle);
        }
    }
}

__global__ void applyGeometryCorrectionCacheCuda(double *m_geometry_cache_d,
                                                 double *m_parameters_d,
                                                 uint16_t *frame) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    int i = threadPosition % (int)m_parameters_d[0];
    int j = threadPosition / (int)m_parameters_d[0];

    if (i >= 0 && i < m_parameters_d[0]) {
        if (j >= 0 && j < m_parameters_d[1]) {
            // TO BE IMPLEMENTED
        }
    }
}

__global__ void buildDepthCorrectionCacheCuda(uint16_t *m_depth_cache_d,
                                              double *m_parameters_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;
    if (threadPosition < m_parameters_d[13]) {
        int16_t currentValue = static_cast<int16_t>(
            static_cast<float>(threadPosition) * m_parameters_d[11] +
            m_parameters_d[12]);
        m_depth_cache_d[threadPosition] = currentValue <= m_parameters_d[14]
                                              ? currentValue
                                              : m_parameters_d[14];
    }
}

__global__ void applyDepthCorrectionCacheCuda(uint16_t* m_frame_d, double* m_parameters_d, uint16_t* m_depth_cache_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    if(threadPosition >=0 && threadPosition < m_parameters_d[0]*m_parameters_d[1])
    {
        *(m_frame_d + threadPosition) = *(m_depth_cache_d + *(m_frame_d + threadPosition));
    }
}

//--------------------------    CLASS   -------------------------------------------------------------

void cudaOnTarget::buildDistortionCorrectionCache() {

    std::cout << "CUDA_CXX: Building Distortion correction\n";

    m_distortion_cache =
        (double *)malloc(sizeof(double) * m_parameters[0] * m_parameters[1]);
    hipMalloc((void **)&m_distortion_cache_d,
               sizeof(double) * m_parameters[0] * m_parameters[1]);

    buildDistortionCorrectionCacheCuda<<<m_parameters[0] * m_parameters[1] /
                                             THREAD_PER_BLOCK,
                                         THREAD_PER_BLOCK>>>(
        m_distortion_cache_d, m_parameters_d);
    hipMemcpy(m_distortion_cache, m_distortion_cache_d,
               sizeof(double) * m_parameters[0] * m_parameters[1],
               hipMemcpyDeviceToHost);

    // std::cout << "GPU distortion: \n";
    // for (int i = 0; i < 10; i++) {
    //     std::cout << m_distortion_cache[i] << ", ";
    // }
    // std::cout << "\n\n\n";
}

void cudaOnTarget::buildGeometryCorrectionCache() {

    std::cout << "CUDA_CXX: Building Geometry correction\n";

    m_geometry_cache =
        (double *)malloc(sizeof(double) * m_parameters[0] * m_parameters[1]);
    hipMalloc((void **)&m_geometry_cache_d,
               sizeof(double) * m_parameters[0] * m_parameters[1]);

    //Check if more blocks nedded than resulted from division
    int nrOfBlocks =
        ((m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK) *
             THREAD_PER_BLOCK <
         m_parameters[0] * m_parameters[1])
            ? m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK + 1
            : m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK;
    buildGeometryCorrectionCacheCuda<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
        m_geometry_cache_d, m_parameters_d);
    hipMemcpy(m_geometry_cache, m_geometry_cache_d,
               sizeof(double) * m_parameters[0] * m_parameters[1],
               hipMemcpyDeviceToHost);

    // std::cout << "GPU geometry: \n";
    // for (int i = 0; i < 10; i++) {
    //     std::cout << m_geometry_cache[i] << ", ";
    // }
    // std::cout << "\n\n\n";
}

void cudaOnTarget::buildDepthCorrectionCache() {

    std::cout << "CUDA_CXX: Building Depth correction\n";

    m_depth_cache = (uint16_t *)malloc(sizeof(uint16_t) * m_parameters[13]);
    hipMalloc((void **)&m_depth_cache_d, sizeof(uint16_t) * m_parameters[13]);

    //Check if more blocks nedded than resulted from division
    int nrOfBlocks = ((m_parameters[13] / THREAD_PER_BLOCK) * THREAD_PER_BLOCK <
                      m_parameters[13])
                         ? m_parameters[13] / THREAD_PER_BLOCK + 1
                         : m_parameters[13] / THREAD_PER_BLOCK;
    buildDepthCorrectionCacheCuda<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
        m_depth_cache_d, m_parameters_d);
    hipMemcpy(m_depth_cache, m_depth_cache_d,
               sizeof(uint16_t) * m_parameters[13], hipMemcpyDeviceToHost);

    // std::cout << "GPU depth: \n";
    // for (int i = 0; i < 10; i++) {
    //     std::cout << m_depth_cache[i] << ", ";
    // }
    // std::cout << "\n\n\n";
}

void cudaOnTarget::applyGeometryCorrection() {}
void cudaOnTarget::applyDistortionCorrection() {}
void cudaOnTarget::applyDepthCorrection() {
    std::cout << "CUDA_CXX: Building Depth correction\n";

    //Check if more blocks nedded than resulted from division
    int nrOfBlocks =
        ((m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK) *
             THREAD_PER_BLOCK <
         m_parameters[0] * m_parameters[1])
            ? m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK + 1
            : m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK;
    applyDepthCorrectionCacheCuda<<<nrOfBlocks, THREAD_PER_BLOCK>>>(m_frame_d, m_parameters_d, m_depth_cache);
}

void cudaOnTarget::cpyFrameToGPU(uint16_t *frame) {
    hipMemcpy(m_frame_d, frame,
               sizeof(uint16_t) * m_parameters[0] * m_parameters[1],
               hipMemcpyHostToDevice);
}
void cudaOnTarget::cpyFrameFromGPU(uint16_t *frame) {
    hipMemcpy(frame, m_frame_d,
               sizeof(uint16_t) * m_parameters[0] * m_parameters[1],
               hipMemcpyDeviceToHost);
}

void cudaOnTarget::setParameters(double width, double height, double fx,
                                 double fy, double cx, double cy, double k1,
                                 double k2, double k3, double x0, double y0,
                                 double gain, double offset,
                                 double pixelMaxValue, double range) {
    //Moving parameters on GPU memory
    double parameters[15] = {width, height, fx, fy, cx,   cy,     k1,
                             k2,    k3,     x0, y0, gain, offset, pixelMaxValue,
                             range};
    m_parameters = (double *)malloc(15 * sizeof(double));
    memcpy(m_parameters, parameters, 15*sizeof(double));

    hipMalloc((void **)&m_parameters_d, sizeof(double) * 15);
    hipMemcpy(m_parameters_d, parameters, sizeof(double) * 15,
               hipMemcpyHostToDevice);

    //allocating memory for frame
    hipMalloc((void **)&m_frame_d, sizeof(uint16_t) * width * height);
}

void cudaOnTarget::freeAll() {
    hipFree(m_geometry_cache_d);
    hipFree(m_distortion_cache_d);
    hipFree(m_depth_cache_d);
    hipFree(m_frame_d);
    hipFree(m_parameters_d);
}